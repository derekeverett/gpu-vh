#include "hip/hip_runtime.h"
/*
 * GhostCells.cu
 *
 *  Created on: Jul 6, 2016
 *      Author: bazow
 */
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "../include/DynamicalVariables.cuh"
#include "../include/GhostCells.cuh"
#include "../include/LatticeParameters.h"
#include "../include/CudaConfiguration.cuh"

void setGhostCells(CONSERVED_VARIABLES * const __restrict__ q,
PRECISION * const __restrict__ e, PRECISION * const __restrict__ p,
FLUID_VELOCITY * const __restrict__ u
) {
	int nstreams = 3;
   hipStream_t *streams = (hipStream_t *) malloc(nstreams * sizeof(hipStream_t));
#pragma unroll 3
   for (int i = 0; i < nstreams; i++) hipStreamCreate(&(streams[i]));

	setGhostCellsKernelI<<<gridSizeGhostI, blockSizeGhostI, 0, streams[0]>>>(q,e,p,u);
	setGhostCellsKernelJ<<<gridSizeGhostJ, blockSizeGhostJ, 0, streams[1]>>>(q,e,p,u);
	setGhostCellsKernelK<<<gridSizeGhostK, blockSizeGhostK, 0, streams[2]>>>(q,e,p,u);
#pragma unroll 3
	for (int i = 0; i < nstreams; i++) hipStreamDestroy(streams[i]);
}

__device__
void setGhostCellVars(CONSERVED_VARIABLES * const __restrict__ q,
PRECISION * const __restrict__ e, PRECISION * const __restrict__ p,
FLUID_VELOCITY * const __restrict__ u,
int s, int sBC) {
	e[s] = e[sBC];
	p[s] = p[sBC];
	u->ut[s] = u->ut[sBC];
	u->ux[s] = u->ux[sBC];
	u->uy[s] = u->uy[sBC];
	u->un[s] = u->un[sBC];
	q->ttt[s] = q->ttt[sBC];
	q->ttx[s] = q->ttx[sBC];
	q->tty[s] = q->tty[sBC];
	q->ttn[s] = q->ttn[sBC];
	// set \pi^\mu\nu ghost cells if evolved
#ifdef PIMUNU
	q->pitt[s] = q->pitt[sBC];
	q->pitx[s] = q->pitx[sBC];
	q->pity[s] = q->pity[sBC];
	q->pitn[s] = q->pitn[sBC];
	q->pixx[s] = q->pixx[sBC];
	q->pixy[s] = q->pixy[sBC];
	q->pixn[s] = q->pixn[sBC];
	q->piyy[s] = q->piyy[sBC];
	q->piyn[s] = q->piyn[sBC];
	q->pinn[s] = q->pinn[sBC];
#endif
	// set \Pi ghost cells if evolved
#ifdef PI
	q->Pi[s] = q->Pi[sBC];
#endif
}

__global__
void setGhostCellsKernelI(CONSERVED_VARIABLES * const __restrict__ q,
PRECISION * const __restrict__ e, PRECISION * const __restrict__ p,
FLUID_VELOCITY * const __restrict__ u
) {
	unsigned int id = blockDim.x * blockIdx.x + threadIdx.x;
	if (id < d_ny*d_nz) {
		unsigned int j = (id % d_ny) + N_GHOST_CELLS_M;
		unsigned int k = id / d_ny + N_GHOST_CELLS_M;

		int iBC = 2;
#pragma unroll 2
		for (int i = 0; i <= 1; ++i) {
			unsigned int s = columnMajorLinearIndex(i, j, k, d_ncx, d_ncy);
			unsigned int sBC = columnMajorLinearIndex(iBC, j, k, d_ncx, d_ncy);
			setGhostCellVars(q,e,p,u,s,sBC);
		}
		iBC = d_nx + 1;
#pragma unroll 2
		for (int i = d_nx + 2; i <= d_nx + 3; ++i) {
			unsigned int s = columnMajorLinearIndex(i, j, k, d_ncx, d_ncy);
			unsigned int sBC = columnMajorLinearIndex(iBC, j, k, d_ncx, d_ncy);
			setGhostCellVars(q,e,p,u,s,sBC);
		}
	}
}

__global__
void setGhostCellsKernelJ(CONSERVED_VARIABLES * const __restrict__ q,
PRECISION * const __restrict__ e, PRECISION * const __restrict__ p,
FLUID_VELOCITY * const __restrict__ u
) {
	unsigned int id = blockDim.x * blockIdx.x + threadIdx.x;
	if (id < d_nx*d_nz) {
		unsigned int i = (id % d_nx) + N_GHOST_CELLS_M;
		unsigned int k = id / d_nx + N_GHOST_CELLS_M;

		int jBC = 2;
#pragma unroll 2
		for (int j = 0; j <= 1; ++j) {
			unsigned int s = columnMajorLinearIndex(i, j, k, d_ncx, d_ncy);
			unsigned int sBC = columnMajorLinearIndex(i, jBC, k, d_ncx, d_ncy);
			setGhostCellVars(q,e,p,u,s,sBC);
		}
		jBC = d_ny + 1;
#pragma unroll 2
		for (int j = d_ny + 2; j <= d_ny + 3; ++j) {
			unsigned int s = columnMajorLinearIndex(i, j, k, d_ncx, d_ncy);
			unsigned int sBC = columnMajorLinearIndex(i, jBC, k, d_ncx, d_ncy);
			setGhostCellVars(q,e,p,u,s,sBC);
		}
	}
}

__global__
void setGhostCellsKernelK(CONSERVED_VARIABLES * const __restrict__ q,
PRECISION * const __restrict__ e, PRECISION * const __restrict__ p,
FLUID_VELOCITY * const __restrict__ u
) {
	unsigned int id = blockDim.x * blockIdx.x + threadIdx.x;
	if (id < d_nx*d_ny) {
		unsigned int i = (id % d_nx) + N_GHOST_CELLS_M;
		unsigned int j = id / d_nx + N_GHOST_CELLS_M;

		int kBC = 2;
#pragma unroll 2
		for (int k = 0; k <= 1; ++k) {
			unsigned int s = columnMajorLinearIndex(i, j, k, d_ncx, d_ncy);
			unsigned int sBC = columnMajorLinearIndex(i, j, kBC, d_ncx, d_ncy);
			setGhostCellVars(q,e,p,u,s,sBC);
		}
		kBC = d_nz + 1;
#pragma unroll 2
		for (int k = d_nz + 2; k <= d_nz + 3; ++k) {
			unsigned int s = columnMajorLinearIndex(i, j, k, d_ncx, d_ncy);
			unsigned int sBC = columnMajorLinearIndex(i, j, kBC, d_ncx, d_ncy);
			setGhostCellVars(q,e,p,u,s,sBC);
		}
	}
}
