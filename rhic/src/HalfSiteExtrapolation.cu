/*
 * HalfSiteExtrapolation.cu
 *
 *  Created on: Oct 22, 2015
 *      Author: bazow
 */

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "../include/HalfSiteExtrapolation.cuh"
#include "../include/FluxLimiter.cuh"
#include "../include/DynamicalVariables.cuh"

__device__
PRECISION rightHalfCellExtrapolationForward(PRECISION qmm, PRECISION qm, PRECISION q, PRECISION qp, PRECISION qpp) {
	return qp - approximateDerivative(q, qp, qpp)/2;
}
__device__
PRECISION rightHalfCellExtrapolationBackwards(PRECISION qmm, PRECISION qm, PRECISION q, PRECISION qp, PRECISION qpp) {
	return q - approximateDerivative(qm, q, qp)/2;
}
__device__
PRECISION leftHalfCellExtrapolationForward(PRECISION qmm, PRECISION qm, PRECISION q, PRECISION qp, PRECISION qpp) {
	return q + approximateDerivative(qm, q, qp)/2;
}
__device__
PRECISION leftHalfCellExtrapolationBackwards(PRECISION qmm, PRECISION qm, PRECISION q, PRECISION qp, PRECISION qpp) {
	return qm + approximateDerivative(qmm, qm, q)/2;
}
