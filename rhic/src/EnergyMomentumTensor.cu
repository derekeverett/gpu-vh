#include "hip/hip_runtime.h"
/*
 * EnergyMomentumTensor.cu
 *
 *  Created on: Oct 22, 2015
 *      Author: bazow
 */
#include <math.h> // for math functions
#include <stdio.h>
#include "../include/EnergyMomentumTensor.cuh"
#include "../include/DynamicalVariables.cuh"
#include "../include/LatticeParameters.h"
#include "../include/CudaConfiguration.cuh"
#include "../include/EquationOfState.cuh"

#define MAX_ITERS 1000000
//const PRECISION ACC = 1e-2;

__host__ __device__
PRECISION energyDensityFromConservedVariables(PRECISION ePrev, PRECISION M0, PRECISION M, PRECISION Pi) {
#ifndef CONFORMAL_EOS
	PRECISION e0 = ePrev;	// initial guess for energy density
	for(int j = 0; j < MAX_ITERS; ++j) {
		PRECISION p = equilibriumPressure(e0);
		PRECISION cs2 = speedOfSoundSquared(e0);
		PRECISION cst2 = p/e0;

		PRECISION A = fmaf(M0,1-cst2,Pi);
		PRECISION B = fmaf(M0,M0+Pi,-M);
		PRECISION H = sqrtf(fabsf(A*A+4*cst2*B));
		PRECISION D = (A-H)/(2*cst2);

		PRECISION f = e0 + D;
		PRECISION fp = 1 - ((cs2 - cst2)*(B + D*H - ((cs2 - cst2)*cst2*D*M0)/e0))/(cst2*e0*H);

		PRECISION converg_factor = 0.9;
		PRECISION e;
		if (j < MAX_ITERS / 2) e = e0 - f/fp;
		else e = e0 - converg_factor * f/fp;
		if(fabsf(e - e0) <=  0.001 * fabsf(e)) return e;
		e0 = e;
	}
//	printf("Maximum number of iterations exceeded.\n");
	printf("Maximum number of iterations exceeded.\tePrev=%.3f,\tM0=%.3f,\t M=%.3f,\t Pi=%.3f \n", ePrev, M0, M, Pi);
	return e0;
#else
	return fabsf(sqrtf(fabsf(4 * M0 * M0 - 3 * M)) - M0);
#endif
}

__host__ __device__
void getInferredVariables(PRECISION t, const PRECISION * const __restrict__ q, PRECISION ePrev,
PRECISION * const __restrict__ e, PRECISION * const __restrict__ p,
PRECISION * const __restrict__ ut, PRECISION * const __restrict__ ux, PRECISION * const __restrict__ uy, PRECISION * const __restrict__ un
) {
	PRECISION ttt = q[0];
	PRECISION ttx = q[1];
	PRECISION tty = q[2];
	PRECISION ttn = q[3];
#ifdef PIMUNU
	PRECISION pitt = q[4];
	PRECISION pitx = q[5];
	PRECISION pity = q[6];
	PRECISION pitn = q[7];
#else
	PRECISION pitt = 0;
	PRECISION pitx = 0;
	PRECISION pity = 0;
	PRECISION pitn = 0;
#endif
	// \Pi
#ifdef PI
	PRECISION Pi = q[14];
#else
	PRECISION Pi = 0;
#endif

	PRECISION M0 = ttt - pitt;
	PRECISION M1 = ttx - pitx;
	PRECISION M2 = tty - pity;
	PRECISION M3 = ttn - pitn;
	PRECISION M = M1 * M1 + M2 * M2 + t * t * M3 * M3;
#ifdef Pi
	if ((M0 * M0 - M + M0 * Pi) < 0)
		Pi = M / M0 - M0;
#endif
/****************************************************************************/
	if (ePrev <= 0.1) {
		*e = M0 - M / M0;
	} else {
		*e = energyDensityFromConservedVariables(ePrev, M0, M, Pi);
		}
	if (isnan(*e)) {
		printf("\n e is nan. M0=%.3f,\t M1=%.3f,\t M2=%.3f,\t M3=%.3f,\t ttt=%.3f,\t ttx=%.3f,\t tty=%.3f,\t ttn=%.3f, \tpitt=%.3f,\t pitx=%.3f,\t pity=%.3f,\t pitn=%.3f\n", M0, M1, M2, M3, ttt, ttx, tty, ttn, pitt, pitx, pity, pitn);
	}
	*p = equilibriumPressure(*e);
	if (*e < 1.e-7) {
		*e = 1.e-7;
		*p = 1.e-7;
	}

	PRECISION P = *p + Pi;
	PRECISION E = 1/(*e + P);
	*ut = sqrtf(fabsf((M0 + P) * E));
	PRECISION E2 = E/(*ut);
	*ux = M1 * E2;
	*uy = M2 * E2;
	*un = M3 * E2;
}

__global__
void setInferredVariablesKernel(const CONSERVED_VARIABLES * const __restrict__ q,
PRECISION * const __restrict__ e, PRECISION * const __restrict__ p, FLUID_VELOCITY * const __restrict__ u,
PRECISION t
) {
	unsigned int threadID = blockDim.x * blockIdx.x + threadIdx.x;

	if (threadID < d_nElements) {
		unsigned int k = threadID / (d_nx * d_ny) + N_GHOST_CELLS_M;
		unsigned int j = (threadID % (d_nx * d_ny)) / d_nx + N_GHOST_CELLS_M;
		unsigned int i = threadID % d_nx + N_GHOST_CELLS_M;
		unsigned int s = columnMajorLinearIndex(i, j, k, d_ncx, d_ncy);

		PRECISION q_s[NUMBER_CONSERVED_VARIABLES];
		q_s[0] = q->ttt[s];
		q_s[1] = q->ttx[s];
		q_s[2] = q->tty[s];
		q_s[3] = q->ttn[s];
#ifdef PIMUNU
		q_s[4] = q->pitt[s];
		q_s[5] = q->pitx[s];
		q_s[6] = q->pity[s];
		q_s[7] = q->pitn[s];
/****************************************************************************\
		q_s[8] = q->pixx[s];
		q_s[9] = q->pixy[s];
		q_s[10] = q->pixn[s];
		q_s[11] = q->piyy[s];
		q_s[12] = q->piyn[s];
		q_s[13] = q->pinn[s];
/****************************************************************************/
#endif
#ifdef PI
		q_s[14] = q->Pi[s];
#endif
		PRECISION _e, _p, ut, ux, uy, un;
		getInferredVariables(t, q_s, e[s], &_e, &_p, &ut, &ux, &uy, &un);
		e[s] = _e;
		p[s] = _p;
		u->ut[s] = ut;
		u->ux[s] = ux;
		u->uy[s] = uy;
		u->un[s] = un;
	}
}

//===================================================================
// Components of T^{\mu\nu} in (\tau,x,y,\eta_s)-coordinates
//===================================================================
__host__ __device__
PRECISION Ttt(PRECISION e, PRECISION p, PRECISION ut, PRECISION pitt) {
	return (e+p)*ut*ut-p+pitt;
}

__host__ __device__
PRECISION Ttx(PRECISION e, PRECISION p, PRECISION ut, PRECISION ux, PRECISION pitx) {
	return (e+p)*ut*ux+pitx;
}

__host__ __device__
PRECISION Tty(PRECISION e, PRECISION p, PRECISION ut, PRECISION uy, PRECISION pity) {
	return (e+p)*ut*uy+pity;
}

__host__ __device__
PRECISION Ttn(PRECISION e, PRECISION p, PRECISION ut, PRECISION un, PRECISION pitn) {
	return (e+p)*ut*un+pitn;
}

__host__ __device__
PRECISION Txx(PRECISION e, PRECISION p, PRECISION ux, PRECISION pixx) {
	return (e+p)*ux*ux+p+pixx;
}

__host__ __device__
PRECISION Txy(PRECISION e, PRECISION p, PRECISION ux, PRECISION uy, PRECISION pixy) {
	return (e+p)*ux*uy+pixy;
}

__host__ __device__
PRECISION Txn(PRECISION e, PRECISION p, PRECISION ux, PRECISION un, PRECISION pixn) {
	return (e+p)*ux*un+pixn;
}

__host__ __device__
PRECISION Tyy(PRECISION e, PRECISION p, PRECISION uy, PRECISION piyy) {
	return (e+p)*uy*uy+p+piyy;
}

__host__ __device__
PRECISION Tyn(PRECISION e, PRECISION p, PRECISION uy, PRECISION un, PRECISION piyn) {
	return (e+p)*uy*un+piyn;
}

__host__ __device__
PRECISION Tnn(PRECISION e, PRECISION p, PRECISION un, PRECISION pinn, PRECISION t) {
	return (e+p)*un*un+p/t/t+pinn;
}
