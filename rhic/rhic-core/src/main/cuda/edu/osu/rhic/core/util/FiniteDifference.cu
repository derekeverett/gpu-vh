/*
 * FiniteDifference.cu
 *
 *  Created on: Oct 22, 2015
 *      Author: bazow
 */

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "edu/osu/rhic/core/util/FiniteDifference.cuh"
#include "edu/osu/rhic/trunk/hydro/DynamicalVariables.cuh"

__device__ 
PRECISION finiteDifferenceX(const PRECISION * const var, int s, int stride, PRECISION fac) {
	return (*(var + s + stride) - *(var + s - stride)) * fac;
}

__device__ 
PRECISION finiteDifferenceY(const PRECISION * const var, int s, int stride, PRECISION fac) {
	return (*(var + s + stride) - *(var + s - stride)) * fac;
}

__device__ 
PRECISION finiteDifferenceZ(const PRECISION * const var, int s, int stride, PRECISION fac) {
	return (*(var + s + stride) - *(var + s - stride)) * fac;
}
