#include "hip/hip_runtime.h"
/*
 * LocalPropagationSpeed.cu
 *
 *  Created on: Oct 22, 2015
 *      Author: bazow
 */

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "edu/osu/rhic/core/muscl/LocalPropagationSpeed.cuh"
#include "edu/osu/rhic/trunk/hydro/DynamicalVariables.cuh"

// maximal local speed at the cell boundaries x_{j\pm 1/2}
__device__ 
PRECISION localPropagationSpeed(PRECISION utr, PRECISION uxr, PRECISION uyr, PRECISION unr,
		PRECISION utl, PRECISION uxl, PRECISION uyl, PRECISION unl,
		PRECISION (*spectralRadius)(PRECISION ut, PRECISION ux, PRECISION uy, PRECISION un)
) {
	PRECISION rhoLeftMovingWave = spectralRadius(utl,uxl,uyl,unl);
	PRECISION rhoRightMovingWave = spectralRadius(utr,uxr,uyr,unr);
	PRECISION a = fmaxf(rhoLeftMovingWave, rhoRightMovingWave);
	return a;
}
