/*
 * DynamicalVariables.cu
 *
 *  Created on: Oct 22, 2015
 *      Author: bazow
 */
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "edu/osu/rhic/trunk/hydro/DynamicalVariables.cuh"
#include "edu/osu/rhic/harness/lattice/LatticeParameters.h"
#include "edu/osu/rhic/trunk/hydro/EnergyMomentumTensor.cuh"
#include "edu/osu/rhic/harness/init/CudaConfiguration.cuh"

CONSERVED_VARIABLES *q;
CONSERVED_VARIABLES *d_q, *d_Q, *d_qS;

FLUID_VELOCITY *u;
FLUID_VELOCITY *d_u, *d_up, *d_uS;

PRECISION *e, *p;
PRECISION *d_e, *d_p;
PRECISION *d_ut, *d_ux, *d_uy, *d_un, *d_utp, *d_uxp, *d_uyp, *d_unp;
PRECISION *d_ttt, *d_ttx, *d_tty, *d_ttn, *d_pitt, *d_pitx, *d_pity, *d_pitn, *d_pixx, *d_pixy, *d_pixn, *d_piyy, *d_piyn, *d_pinn, *d_Pi;

VALIDITY_DOMAIN *validityDomain, *d_validityDomain;
PRECISION *d_regulations, *d_knudsenNumberTaupi, *d_knudsenNumberTauPi, *d_inverseReynoldsNumberPimunu, *d_inverseReynoldsNumber2Pimunu,
		*d_inverseReynoldsNumberTilde2Pimunu, *d_inverseReynoldsNumberPi, *d_inverseReynoldsNumber2Pi, *d_inverseReynoldsNumberTilde2Pi;
// for debugging
PRECISION *d_taupi, *d_dxux, *d_dyuy, *d_theta;

__host__ __device__
int columnMajorLinearIndex(int i, int j, int k, int nx, int ny) {
	return i + nx * (j + ny * k);
}

void allocateHostMemory(int len) {
	size_t bytes = sizeof(PRECISION);
	e = (PRECISION *) calloc(len, bytes);
	p = (PRECISION *) calloc(len, bytes);

	u = (FLUID_VELOCITY *) calloc(1, sizeof(FLUID_VELOCITY));
	u->ut = (PRECISION *) calloc(len, bytes);
	u->ux = (PRECISION *) calloc(len, bytes);
	u->uy = (PRECISION *) calloc(len, bytes);
	u->un = (PRECISION *) calloc(len, bytes);

	q = (CONSERVED_VARIABLES *) calloc(1, sizeof(CONSERVED_VARIABLES));
	q->ttt = (PRECISION *) calloc(len, bytes);
	q->ttx = (PRECISION *) calloc(len, bytes);
	q->tty = (PRECISION *) calloc(len, bytes);
	q->ttn = (PRECISION *) calloc(len, bytes);
	// allocate space for \pi^\mu\nu
#ifdef PIMUNU
	q->pitt = (PRECISION *) calloc(len, bytes);
	q->pitx = (PRECISION *) calloc(len, bytes);
	q->pity = (PRECISION *) calloc(len, bytes);
	q->pitn = (PRECISION *) calloc(len, bytes);
	q->pixx = (PRECISION *) calloc(len, bytes);
	q->pixy = (PRECISION *) calloc(len, bytes);
	q->pixn = (PRECISION *) calloc(len, bytes);
	q->piyy = (PRECISION *) calloc(len, bytes);
	q->piyn = (PRECISION *) calloc(len, bytes);
	q->pinn = (PRECISION *) calloc(len, bytes);
#endif
	// allocate space for \Pi
#ifdef PI
	q->Pi = (PRECISION *) calloc(len, bytes);
#endif

	validityDomain = (VALIDITY_DOMAIN *) calloc(1, sizeof(VALIDITY_DOMAIN));
	validityDomain->regulations = (PRECISION *) calloc(len, bytes);
	validityDomain->knudsenNumberTaupi = (PRECISION *) calloc(len, bytes);
	validityDomain->knudsenNumberTauPi = (PRECISION *) calloc(len, bytes);
	validityDomain->inverseReynoldsNumberPimunu = (PRECISION *) calloc(len, bytes);
	validityDomain->inverseReynoldsNumber2Pimunu = (PRECISION *) calloc(len, bytes);
	validityDomain->inverseReynoldsNumberTilde2Pimunu = (PRECISION *) calloc(len, bytes);
	validityDomain->inverseReynoldsNumberPi = (PRECISION *) calloc(len, bytes);
	validityDomain->inverseReynoldsNumber2Pi = (PRECISION *) calloc(len, bytes);
	validityDomain->inverseReynoldsNumberTilde2Pi = (PRECISION *) calloc(len, bytes);
	for(int s=0; s<len; ++s) validityDomain->regulations[s] = (PRECISION) 1.0;
	validityDomain->taupi = (PRECISION *) calloc(len, bytes);
	validityDomain->dxux = (PRECISION *) calloc(len, bytes);
	validityDomain->dyuy = (PRECISION *) calloc(len, bytes);
	validityDomain->theta = (PRECISION *) calloc(len, bytes);
}

void allocateIntermidateFluidVelocityDeviceMemory(FLUID_VELOCITY *d_u, size_t size2) {
	PRECISION *d_ut, *d_ux, *d_uy, *d_un;
	hipMalloc((void **) &d_ut, size2);
	hipMalloc((void **) &d_ux, size2);
	hipMalloc((void **) &d_uy, size2);
	hipMalloc((void **) &d_un, size2);

	hipMemcpy(&(d_u->ut), &d_ut, sizeof(PRECISION*), hipMemcpyHostToDevice);
	hipMemcpy(&(d_u->ux), &d_ux, sizeof(PRECISION*), hipMemcpyHostToDevice);
	hipMemcpy(&(d_u->uy), &d_uy, sizeof(PRECISION*), hipMemcpyHostToDevice);
	hipMemcpy(&(d_u->un), &d_un, sizeof(PRECISION*), hipMemcpyHostToDevice);
}

void allocateIntermidateConservedVarDeviceMemory(CONSERVED_VARIABLES *d_q, size_t bytes) {
	//=======================================================
	// Conserved variables
	//=======================================================
	PRECISION *d_ttt, *d_ttx, *d_tty, *d_ttn, *d_pitt, *d_pitx, *d_pity, *d_pitn, *d_pixx, *d_pixy, *d_pixn, *d_piyy, *d_piyn, *d_pinn, *d_Pi;
	hipMalloc((void **) &d_ttt, bytes);
	hipMalloc((void **) &d_ttx, bytes);
	hipMalloc((void **) &d_tty, bytes);
	hipMalloc((void **) &d_ttn, bytes);

	hipMemcpy(&(d_q->ttt), &d_ttt, sizeof(PRECISION*), hipMemcpyHostToDevice);
	hipMemcpy(&(d_q->ttx), &d_ttx, sizeof(PRECISION*), hipMemcpyHostToDevice);
	hipMemcpy(&(d_q->tty), &d_tty, sizeof(PRECISION*), hipMemcpyHostToDevice);
	hipMemcpy(&(d_q->ttn), &d_ttn, sizeof(PRECISION*), hipMemcpyHostToDevice);
	// allocate space for \pi^\mu\nu
#ifdef PIMUNU
	hipMalloc((void **) &d_pitt, bytes);
	hipMalloc((void **) &d_pitx, bytes);
	hipMalloc((void **) &d_pity, bytes);
	hipMalloc((void **) &d_pitn, bytes);
	hipMalloc((void **) &d_pixx, bytes);
	hipMalloc((void **) &d_pixy, bytes);
	hipMalloc((void **) &d_pixn, bytes);
	hipMalloc((void **) &d_piyy, bytes);
	hipMalloc((void **) &d_piyn, bytes);
	hipMalloc((void **) &d_pinn, bytes);

	hipMemcpy(&(d_q->pitt), &d_pitt, sizeof(PRECISION*), hipMemcpyHostToDevice);
	hipMemcpy(&(d_q->pitx), &d_pitx, sizeof(PRECISION*), hipMemcpyHostToDevice);
	hipMemcpy(&(d_q->pity), &d_pity, sizeof(PRECISION*), hipMemcpyHostToDevice);
	hipMemcpy(&(d_q->pitn), &d_pitn, sizeof(PRECISION*), hipMemcpyHostToDevice);
	hipMemcpy(&(d_q->pixx), &d_pixx, sizeof(PRECISION*), hipMemcpyHostToDevice);
	hipMemcpy(&(d_q->pixy), &d_pixy, sizeof(PRECISION*), hipMemcpyHostToDevice);
	hipMemcpy(&(d_q->pixn), &d_pixn, sizeof(PRECISION*), hipMemcpyHostToDevice);
	hipMemcpy(&(d_q->piyy), &d_piyy, sizeof(PRECISION*), hipMemcpyHostToDevice);
	hipMemcpy(&(d_q->piyn), &d_piyn, sizeof(PRECISION*), hipMemcpyHostToDevice);
	hipMemcpy(&(d_q->pinn), &d_pinn, sizeof(PRECISION*), hipMemcpyHostToDevice);
#endif
	// allocate space for \Pi
#ifdef PI
	hipMalloc((void **) &d_Pi, bytes);

	hipMemcpy(&(d_q->Pi), &d_Pi, sizeof(PRECISION*), hipMemcpyHostToDevice);
#endif
}

void allocateDeviceMemory(size_t bytes) {
	//=======================================================
	// Energy density and pressure
	//=======================================================	
	hipMalloc((void **) &d_e, bytes);
	hipMalloc((void **) &d_p, bytes);

	//=======================================================
	// Fluid velocity
	//=======================================================				
	hipMalloc((void **) &d_ut, bytes);
	hipMalloc((void **) &d_ux, bytes);
	hipMalloc((void **) &d_uy, bytes);
	hipMalloc((void **) &d_un, bytes);
	hipMalloc((void **) &d_utp, bytes);
	hipMalloc((void **) &d_uxp, bytes);
	hipMalloc((void **) &d_uyp, bytes);
	hipMalloc((void **) &d_unp, bytes);

	hipMalloc((void**) &d_u, sizeof(FLUID_VELOCITY));
	hipMalloc((void**) &d_up, sizeof(FLUID_VELOCITY));
	hipMalloc((void**) &d_uS, sizeof(FLUID_VELOCITY));

	hipMemcpy(&(d_u->ut), &d_ut, sizeof(PRECISION*), hipMemcpyHostToDevice);
	hipMemcpy(&(d_u->ux), &d_ux, sizeof(PRECISION*), hipMemcpyHostToDevice);
	hipMemcpy(&(d_u->uy), &d_uy, sizeof(PRECISION*), hipMemcpyHostToDevice);
	hipMemcpy(&(d_u->un), &d_un, sizeof(PRECISION*), hipMemcpyHostToDevice);

	hipMemcpy(&(d_up->ut), &d_utp, sizeof(PRECISION*), hipMemcpyHostToDevice);
	hipMemcpy(&(d_up->ux), &d_uxp, sizeof(PRECISION*), hipMemcpyHostToDevice);
	hipMemcpy(&(d_up->uy), &d_uyp, sizeof(PRECISION*), hipMemcpyHostToDevice);
	hipMemcpy(&(d_up->un), &d_unp, sizeof(PRECISION*), hipMemcpyHostToDevice);

	//=======================================================
	// Conserved variables
	//=======================================================
	hipMalloc((void **) &d_ttt, bytes);
	hipMalloc((void **) &d_ttx, bytes);
	hipMalloc((void **) &d_tty, bytes);
	hipMalloc((void **) &d_ttn, bytes);

	hipMalloc((void**) &d_q, sizeof(CONSERVED_VARIABLES));
	hipMalloc((void**) &d_Q, sizeof(CONSERVED_VARIABLES));
	hipMalloc((void**) &d_qS, sizeof(CONSERVED_VARIABLES));

	hipMemcpy(&(d_q->ttt), &d_ttt, sizeof(PRECISION*), hipMemcpyHostToDevice);
	hipMemcpy(&(d_q->ttx), &d_ttx, sizeof(PRECISION*), hipMemcpyHostToDevice);
	hipMemcpy(&(d_q->tty), &d_tty, sizeof(PRECISION*), hipMemcpyHostToDevice);
	hipMemcpy(&(d_q->ttn), &d_ttn, sizeof(PRECISION*), hipMemcpyHostToDevice);

	// allocate space for \pi^\mu\nu
#ifdef PIMUNU
	hipMalloc((void **) &d_pitt, bytes);
	hipMalloc((void **) &d_pitx, bytes);
	hipMalloc((void **) &d_pity, bytes);
	hipMalloc((void **) &d_pitn, bytes);
	hipMalloc((void **) &d_pixx, bytes);
	hipMalloc((void **) &d_pixy, bytes);
	hipMalloc((void **) &d_pixn, bytes);
	hipMalloc((void **) &d_piyy, bytes);
	hipMalloc((void **) &d_piyn, bytes);
	hipMalloc((void **) &d_pinn, bytes);

	hipMemcpy(&(d_q->pitt), &d_pitt, sizeof(PRECISION*), hipMemcpyHostToDevice);
	hipMemcpy(&(d_q->pitx), &d_pitx, sizeof(PRECISION*), hipMemcpyHostToDevice);
	hipMemcpy(&(d_q->pity), &d_pity, sizeof(PRECISION*), hipMemcpyHostToDevice);
	hipMemcpy(&(d_q->pitn), &d_pitn, sizeof(PRECISION*), hipMemcpyHostToDevice);
	hipMemcpy(&(d_q->pixx), &d_pixx, sizeof(PRECISION*), hipMemcpyHostToDevice);
	hipMemcpy(&(d_q->pixy), &d_pixy, sizeof(PRECISION*), hipMemcpyHostToDevice);
	hipMemcpy(&(d_q->pixn), &d_pixn, sizeof(PRECISION*), hipMemcpyHostToDevice);
	hipMemcpy(&(d_q->piyy), &d_piyy, sizeof(PRECISION*), hipMemcpyHostToDevice);
	hipMemcpy(&(d_q->piyn), &d_piyn, sizeof(PRECISION*), hipMemcpyHostToDevice);
	hipMemcpy(&(d_q->pinn), &d_pinn, sizeof(PRECISION*), hipMemcpyHostToDevice);
#endif
	// allocate space for \Pi
#ifdef PI
	hipMalloc((void **) &d_Pi, bytes);

	hipMemcpy(&(d_q->Pi), &d_Pi, sizeof(PRECISION*), hipMemcpyHostToDevice);
#endif

	//=======================================================
	// Intermediate fluid velocity/conserved variables
	//=======================================================
	allocateIntermidateConservedVarDeviceMemory(d_Q, bytes);
	allocateIntermidateConservedVarDeviceMemory(d_qS, bytes);
	allocateIntermidateFluidVelocityDeviceMemory(d_uS, bytes);

	//=======================================================
	// Hydrodynamic validity
	//=======================================================
	hipMalloc((void **) &d_regulations, bytes);
	hipMalloc((void **) &d_knudsenNumberTaupi, bytes);
	hipMalloc((void **) &d_knudsenNumberTauPi, bytes);
	hipMalloc((void **) &d_inverseReynoldsNumberPimunu, bytes);
	hipMalloc((void **) &d_inverseReynoldsNumber2Pimunu, bytes);
	hipMalloc((void **) &d_inverseReynoldsNumberTilde2Pimunu, bytes);
	hipMalloc((void **) &d_inverseReynoldsNumberPi, bytes);
	hipMalloc((void **) &d_inverseReynoldsNumber2Pi, bytes);
	hipMalloc((void **) &d_inverseReynoldsNumberTilde2Pi, bytes);
	// for debugging purposes
	hipMalloc((void **) &d_taupi, bytes);
	hipMalloc((void **) &d_dxux, bytes);
	hipMalloc((void **) &d_dyuy, bytes);
	hipMalloc((void **) &d_theta, bytes);

	hipMalloc((void**) &d_validityDomain, sizeof(VALIDITY_DOMAIN));

	hipMemcpy(&(d_validityDomain->regulations), &d_regulations, sizeof(PRECISION*), hipMemcpyHostToDevice);
	hipMemcpy(&(d_validityDomain->knudsenNumberTaupi), &d_knudsenNumberTaupi, sizeof(PRECISION*), hipMemcpyHostToDevice);
	hipMemcpy(&(d_validityDomain->knudsenNumberTauPi), &d_knudsenNumberTauPi, sizeof(PRECISION*), hipMemcpyHostToDevice);
	hipMemcpy(&(d_validityDomain->inverseReynoldsNumberPimunu), &d_inverseReynoldsNumberPimunu, sizeof(PRECISION*), hipMemcpyHostToDevice);
	hipMemcpy(&(d_validityDomain->inverseReynoldsNumber2Pimunu), &d_inverseReynoldsNumber2Pimunu, sizeof(PRECISION*), hipMemcpyHostToDevice);
	hipMemcpy(&(d_validityDomain->inverseReynoldsNumberTilde2Pimunu), &d_inverseReynoldsNumberTilde2Pimunu, sizeof(PRECISION*), hipMemcpyHostToDevice);
	hipMemcpy(&(d_validityDomain->inverseReynoldsNumberPi), &d_inverseReynoldsNumberPi, sizeof(PRECISION*), hipMemcpyHostToDevice);
	hipMemcpy(&(d_validityDomain->inverseReynoldsNumber2Pi), &d_inverseReynoldsNumber2Pi, sizeof(PRECISION*), hipMemcpyHostToDevice);
	hipMemcpy(&(d_validityDomain->inverseReynoldsNumberTilde2Pi), &d_inverseReynoldsNumberTilde2Pi, sizeof(PRECISION*), hipMemcpyHostToDevice);
	// for debugging
	hipMemcpy(&(d_validityDomain->taupi), &d_taupi, sizeof(PRECISION*), hipMemcpyHostToDevice);
	hipMemcpy(&(d_validityDomain->dxux), &d_dxux, sizeof(PRECISION*), hipMemcpyHostToDevice);
	hipMemcpy(&(d_validityDomain->dyuy), &d_dyuy, sizeof(PRECISION*), hipMemcpyHostToDevice);
	hipMemcpy(&(d_validityDomain->theta), &d_theta, sizeof(PRECISION*), hipMemcpyHostToDevice);
}

void copyHostToDeviceMemory(size_t bytes) {
	hipMemcpy(d_e, e, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_p, p, bytes, hipMemcpyHostToDevice);

	hipMemcpy(d_ut, u->ut, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_ux, u->ux, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_uy, u->uy, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_un, u->un, bytes, hipMemcpyHostToDevice);

	//Also initialize the value of up on the device!
	hipMemcpy(d_utp, u->ut, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_uxp, u->ux, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_uyp, u->uy, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_unp, u->un, bytes, hipMemcpyHostToDevice);

	hipMemcpy(d_ttt, q->ttt, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_ttx, q->ttx, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_tty, q->tty, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_ttn, q->ttn, bytes, hipMemcpyHostToDevice);
	// copy \pi^\mu\nu to device memory
#ifdef PIMUNU
	hipMemcpy(d_pitt, q->pitt, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_pitx, q->pitx, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_pity, q->pity, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_pitn, q->pitn, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_pixx, q->pixx, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_pixy, q->pixy, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_pixn, q->pixn, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_piyy, q->piyy, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_piyn, q->piyn, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_pinn, q->pinn, bytes, hipMemcpyHostToDevice);
#endif
	// copy \Pi to device memory
#ifdef PI
	hipMemcpy(d_Pi, q->Pi, bytes, hipMemcpyHostToDevice);
#endif
	hipMemcpy(d_regulations, validityDomain->regulations, bytes, hipMemcpyHostToDevice);
}

void copyDeviceToHostMemory(size_t bytes) {
	hipMemcpy(e, d_e, bytes, hipMemcpyDeviceToHost);
	hipMemcpy(p, d_p, bytes, hipMemcpyDeviceToHost);
	hipMemcpy(u->ut, d_ut, bytes, hipMemcpyDeviceToHost);
	hipMemcpy(u->ux, d_ux, bytes, hipMemcpyDeviceToHost);
	hipMemcpy(u->uy, d_uy, bytes, hipMemcpyDeviceToHost);
	hipMemcpy(u->un, d_un, bytes, hipMemcpyDeviceToHost);
#ifdef PIMUNU
	hipMemcpy(q->pitt, d_pitt, bytes, hipMemcpyDeviceToHost);
	hipMemcpy(q->pitx, d_pitx, bytes, hipMemcpyDeviceToHost);
	hipMemcpy(q->pity, d_pity, bytes, hipMemcpyDeviceToHost);
	hipMemcpy(q->pitn, d_pitn, bytes, hipMemcpyDeviceToHost);
	hipMemcpy(q->pixx, d_pixx, bytes, hipMemcpyDeviceToHost);
	hipMemcpy(q->pixy, d_pixy, bytes, hipMemcpyDeviceToHost);
	hipMemcpy(q->pixn, d_pixn, bytes, hipMemcpyDeviceToHost);
	hipMemcpy(q->piyy, d_piyy, bytes, hipMemcpyDeviceToHost);
	hipMemcpy(q->piyn, d_piyn, bytes, hipMemcpyDeviceToHost);
	hipMemcpy(q->pinn, d_pinn, bytes, hipMemcpyDeviceToHost);
#endif
#ifdef PI
	hipMemcpy(q->Pi, d_Pi, bytes, hipMemcpyDeviceToHost);
#endif

	hipMemcpy(validityDomain->regulations, d_regulations, bytes, hipMemcpyDeviceToHost);
	hipMemcpy(validityDomain->knudsenNumberTaupi, d_knudsenNumberTaupi, bytes, hipMemcpyDeviceToHost);
	hipMemcpy(validityDomain->knudsenNumberTauPi, d_knudsenNumberTauPi, bytes, hipMemcpyDeviceToHost);
	hipMemcpy(validityDomain->inverseReynoldsNumberPimunu, d_inverseReynoldsNumberPimunu, bytes, hipMemcpyDeviceToHost);
	hipMemcpy(validityDomain->inverseReynoldsNumber2Pimunu, d_inverseReynoldsNumber2Pimunu, bytes, hipMemcpyDeviceToHost);
	hipMemcpy(validityDomain->inverseReynoldsNumberTilde2Pimunu, d_inverseReynoldsNumberTilde2Pimunu, bytes, hipMemcpyDeviceToHost);
	hipMemcpy(validityDomain->inverseReynoldsNumberPi, d_inverseReynoldsNumberPi, bytes, hipMemcpyDeviceToHost);
	hipMemcpy(validityDomain->inverseReynoldsNumber2Pi, d_inverseReynoldsNumber2Pi, bytes, hipMemcpyDeviceToHost);
	hipMemcpy(validityDomain->inverseReynoldsNumberTilde2Pi, d_inverseReynoldsNumberTilde2Pi, bytes, hipMemcpyDeviceToHost);
	// for debugging
	hipMemcpy(validityDomain->taupi, d_taupi, bytes, hipMemcpyDeviceToHost);
	hipMemcpy(validityDomain->dxux, d_dxux, bytes, hipMemcpyDeviceToHost);
	hipMemcpy(validityDomain->dyuy, d_dyuy, bytes, hipMemcpyDeviceToHost);
	hipMemcpy(validityDomain->theta, d_theta, bytes, hipMemcpyDeviceToHost);
}

void setConservedVariables(double t, void * latticeParams) {
	struct LatticeParameters * lattice = (struct LatticeParameters *) latticeParams;

	int nx = lattice->numLatticePointsX;
	int ny = lattice->numLatticePointsY;
	int nz = lattice->numLatticePointsRapidity;
	int ncx = lattice->numComputationalLatticePointsX;
	int ncy = lattice->numComputationalLatticePointsY;

	for (int k = N_GHOST_CELLS_M; k < nz + N_GHOST_CELLS_M; ++k) {
		for (int j = N_GHOST_CELLS_M; j < ny + N_GHOST_CELLS_M; ++j) {
			for (int i = N_GHOST_CELLS_M; i < nx + N_GHOST_CELLS_M; ++i) {
				int s = columnMajorLinearIndex(i, j, k, ncx, ncy);

				PRECISION ux_s = u->ux[s];
				PRECISION uy_s = u->uy[s];
				PRECISION un_s = u->un[s];
				PRECISION ut_s = u->ut[s];
				PRECISION e_s = e[s];
				PRECISION p_s = p[s];

				PRECISION pitt_s = 0;
				PRECISION pitx_s = 0;
				PRECISION pity_s = 0;
				PRECISION pitn_s = 0;
#ifdef PIMUNU
				pitt_s = q->pitt[s];
				pitx_s = q->pitx[s];
				pity_s = q->pity[s];
				pitn_s = q->pitn[s];
#endif
				PRECISION Pi_s = 0;
#ifdef PI
				Pi_s = q->Pi[s];
#endif

				q->ttt[s] = Ttt(e_s, p_s + Pi_s, ut_s, pitt_s);
				q->ttx[s] = Ttx(e_s, p_s + Pi_s, ut_s, ux_s, pitx_s);
				q->tty[s] = Tty(e_s, p_s + Pi_s, ut_s, uy_s, pity_s);
				q->ttn[s] = Ttn(e_s, p_s + Pi_s, ut_s, un_s, pitn_s);
			}
		}
	}
}

void swap(CONSERVED_VARIABLES **arr1, CONSERVED_VARIABLES **arr2) {
	CONSERVED_VARIABLES *tmp = *arr1;
	*arr1 = *arr2;
	*arr2 = tmp;
}

void setCurrentConservedVariables() {
	swap(&d_q, &d_Q);
}

void swapFluidVelocity(FLUID_VELOCITY **arr1, FLUID_VELOCITY **arr2) {
	FLUID_VELOCITY *tmp = *arr1;
	*arr1 = *arr2;
	*arr2 = tmp;
}

void freeHostMemory() {
	free(e);
	free(p);
	free(u->ut);
	free(u->ux);
	free(u->uy);
	free(u->un);
	free(u);

	free(q->ttt);
	free(q->ttx);
	free(q->tty);
	free(q->ttn);
	// free \pi^\mu\nu
#ifdef PIMUNU
	free(q->pitt);
	free(q->pitx);
	free(q->pity);
	free(q->pitn);
	free(q->pixx);
	free(q->pixy);
	free(q->pixn);
	free(q->piyy);
	free(q->piyn);
	free(q->pinn);
#endif
	// free \Pi
#ifdef PI
	free(q->Pi);
#endif
	free(q);
}

void freeDeviceMemory() {
	hipFree(d_e);
	hipFree(d_p);

	hipFree(d_ut);
	hipFree(d_ux);
	hipFree(d_uy);
	hipFree(d_un);
	hipFree(d_utp);
	hipFree(d_uxp);
	hipFree(d_uyp);
	hipFree(d_unp);

	hipFree(d_u);
	hipFree(d_up);
	hipFree(d_uS);

	hipFree(d_ttt);
	hipFree(d_ttx);
	hipFree(d_tty);
	hipFree(d_ttn);
	// free \pi^\mu\nu
#ifdef PIMUNU
	hipFree(d_pitt);
	hipFree(d_pitx);
	hipFree(d_pity);
	hipFree(d_pitn);
	hipFree(d_pixx);
	hipFree(d_pixy);
	hipFree(d_pixn);
	hipFree(d_piyy);
	hipFree(d_piyn);
	hipFree(d_pinn);
#endif
	// free \Pi
#ifdef PI
	hipFree(d_Pi);
#endif

	hipFree(d_q);
	hipFree(d_Q);
	hipFree(d_qS);
}
