/*
 * FluxFunctions.cu
 *
 *  Created on: Oct 22, 2015
 *      Author: bazow
 */

#include <stdlib.h>
#include <stdio.h> // for printf

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "edu/osu/rhic/trunk/hydro/FluxFunctions.cuh"
#include "edu/osu/rhic/trunk/hydro/EnergyMomentumTensor.cuh"
#include "edu/osu/rhic/trunk/hydro/DynamicalVariables.cuh"

__device__ 
PRECISION Fx(PRECISION q, PRECISION ut, PRECISION ux, PRECISION uy, PRECISION un) {
	return ux * q / ut;
}

__device__ 
PRECISION Fy(PRECISION q, PRECISION ut, PRECISION ux, PRECISION uy, PRECISION un) {
	return uy * q / ut;
}

__device__ 
PRECISION Fz(PRECISION q, PRECISION ut, PRECISION ux, PRECISION uy, PRECISION un) {
	return un * q / ut;
}
