#include "hip/hip_runtime.h"
/*
 * SpectralRadius.cu
 *
 *  Created on: Oct 22, 2015
 *      Author: bazow
 */

#include <math.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "edu/osu/rhic/trunk/hydro/SpectralRadius.cuh"
#include "edu/osu/rhic/trunk/hydro/EnergyMomentumTensor.cuh"
#include "edu/osu/rhic/trunk/hydro/DynamicalVariables.cuh"

__device__ 
PRECISION spectralRadiusX(PRECISION ut, PRECISION ux, PRECISION uy, PRECISION un) {
	return fabsf(ux/ut);
}

__device__ 
PRECISION spectralRadiusY(PRECISION ut, PRECISION ux, PRECISION uy, PRECISION un) {
	return fabsf(uy/ut);
}

__device__ 
PRECISION spectralRadiusZ(PRECISION ut, PRECISION ux, PRECISION uy, PRECISION un) {
	return fabsf(un/ut);
}
